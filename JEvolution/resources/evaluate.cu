
#include <hip/hip_runtime.h>
extern "C"
__global__ void find_reduce_point(char *nodes, int node_count, int count, int threads, int blocks,  int *out)
{
	char TERMINAL = 0;
	char NETERMINAL = 1;
	char NONE = 0xf;
	
	int TYPE_INDEX = 0;
	int MODEL_INDEX = 1;
	int CHILD_INDEX = 2;
	int VALUE_INDEX = 3;
	
	int NODE_SIZE = 11;
	
	int node_offset = count * NODE_SIZE;
	int tid = blockIdx.x * threads + threadIdx.x;
	
	while (tid < count) {
		int start_pos = tid * NODE_SIZE;
		int child = 0;
		int actual = start_pos;
		
		if (nodes[start_pos + TYPE_INDEX] == TERMINAL) {
			out[tid] = 0;
		} 

		for (int i = start_pos + node_offset; i < (node_offset * node_count); i += node_offset) {
			if (nodes[i + TYPE_INDEX] == TERMINAL)
				child++;
			else if (nodes[i + TYPE_INDEX] == NETERMINAL) {
				child = 0;
				actual = i;
			} else {
				continue;
			}	

			if (child == nodes[actual + CHILD_INDEX]) {
				out[tid] = (actual / NODE_SIZE / count);
				break;	
			}
		}
		
		tid += threads * blocks;
	}
	__syncthreads();
}